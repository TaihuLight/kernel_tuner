#include "hip/hip_runtime.h"
#define image_height 4096
#define image_width 4096
#define filter_height 17
#define filter_width 17

#define border_height ((filter_height/2)*2)
#define border_width ((filter_width/2)*2)
#define input_height (image_height + border_height)
#define input_width (image_width + border_width)

#define i_end min(block_size_y*tile_size_y+border_height, input_height)
#define j_end min(block_size_x*tile_size_x+border_width, input_width)

__constant__ float d_filter[filter_height*filter_width];

__global__ void convolution_kernel(float *output, float *input, float *filter) {
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int by = blockIdx.y * block_size_y * tile_size_y;
    int bx = blockIdx.x * block_size_x * tile_size_x;

    //shared memory to hold all input data need by this thread block
    __shared__ float sh_input[block_size_y*tile_size_y+border_height][block_size_x*tile_size_x+border_width];

    //load all input data needed by this thread block into shared memory
    #pragma unroll
    for (int i=ty; i<i_end; i+=block_size_y) {
        #pragma unroll
        for (int j=tx; j<j_end; j+=block_size_x) {
            #if ((image_height%(block_size_y*tile_size_y)!=0) || (image_width%(block_size_x*tile_size_x)!=0))
            int y = by+i;
            int x = bx+j;
            if (y < input_height && x < input_width) {
                sh_input[i][j] = input[y*input_width+x];
            }
            #else
                sh_input[i][j] = input[(by+i)*input_width + (bx+j)];
            #endif
        }
    }
    __syncthreads();

    //thread-local registers to hold local sums
    float sum[tile_size_y][tile_size_x];
    #pragma unroll
    for (int yi=0; yi<tile_size_y; yi++) {
        #pragma unroll
        for (int xi=0; xi<tile_size_x; xi++) {
             sum[yi][xi] = 0.0f;
        }
    }

    //for each filter weight
    #pragma unroll
    for (int i=0; i < filter_height; i++) {
        #pragma unroll
        for (int j=0; j < filter_width; j++) {

            #pragma unroll
            for (int yi=0; yi<tile_size_y; yi++) {   
                #pragma unroll
                for (int xi=0; xi<tile_size_x; xi++) {
                    sum[yi][xi] += sh_input[ty+yi*block_size_y+i][tx+xi*block_size_x+j] * d_filter[i*filter_width+j];
                }
            }

        }
    }

    //store results to global memory
    #pragma unroll
    for (int yi=0; yi<tile_size_y; yi++) {   
        #pragma unroll
        for (int xi=0; xi<tile_size_x; xi++) {
            #if ((image_height%(block_size_y*tile_size_y)!=0) || (image_width%(block_size_x*tile_size_x)!=0))
            int y = by+ty+yi*block_size_y;
            int x = bx+tx+xi*block_size_x;
            if (y < image_height && x < image_width) {
                output[y * image_width + x] = sum[yi][xi];
            }
            #else
                output[(by+ty+yi*block_size_y) * image_width + bx+tx+xi*block_size_x] = sum[yi][xi];
            #endif
        }
    }

}





__global__ void convolution_naive(float *output, float *input, float *filter) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int i, j;
    float sum = 0.0;

    if (y < image_height && x < image_width) {

        for (j = 0; j < filter_height; j++) {
            for (i = 0; i < filter_width; i++) {
                sum += input[(y + j) * input_width + (x + i)] * filter[j * filter_width + i];
            }
        }

        output[y * image_width + x] = sum;
    }
}
